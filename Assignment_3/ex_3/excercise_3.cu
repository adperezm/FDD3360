
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Question 1 defines:
#define FIXED_DT 1
#define BLOCK_SIZE 64
#define VEL 1
typedef float3 Particle;


//function declarations
__device__ inline float3& operator +=(float3 &a, const float3 &b);
__global__ void GPU_update(Particle *all_particles_ptr, float v,unsigned long long offset,unsigned int streamsize,unsigned int num_iterations); //Computes the update of only one update for all particles.
void particle_computation_launcher(unsigned int n_streams, unsigned long long num_particles,unsigned int batch_size,unsigned int num_iterations);
//function main
int main(int argc, char *argv[])
{
    printf("Running! \n");   
    unsigned int n_streams = atoi(argv[1]); //1 indicates the number of strings to use
    unsigned long long num_particles = ((unsigned long long)atoi(argv[2]))*1000; //The number of thousands of particles to use
    //unsigned int batch_size= std::max(1000,num_particles/2); //We use 2 batches if num particles is short. Else 1000 threads per batch, so we use most of the cores. I should try with different values of this to see optimal
    unsigned int batch_size= atoi(argv[3]);
    unsigned int num_iterations=atoi(argv[4]);
    particle_computation_launcher(n_streams, num_particles,batch_size,num_iterations);  

    return 0;
}

//function definitions
void particle_computation_launcher(unsigned int n_streams, unsigned long long num_particles,unsigned int batch_size, unsigned int num_iterations)
{
    float v=VEL; //Could be a random value.
    Particle *d_all_particles_ptr;
    Particle *all_particles_ptr;
    hipStream_t *streams;
    const unsigned int streamSize=batch_size;
    const unsigned int streamBytes=streamSize*sizeof(Particle);
    if(num_particles%(batch_size) !=0)
    {
        fprintf(stderr, "Only programmed for num particles a multiple of n streams");
        exit(1);
    }
    //Allocates memory in host, device, and sets the values to 0.
    hipMalloc(&d_all_particles_ptr, num_particles*sizeof(Particle)); 
    if (hipHostMalloc(&all_particles_ptr, num_particles*sizeof(Particle), hipHostMallocDefault) != hipSuccess)
	    {printf("Could not allocate pag memory!");};
    for(unsigned int i=0; i< num_particles;i++)
    {
        all_particles_ptr[i].x=0;
        all_particles_ptr[i].y=0;
        all_particles_ptr[i].z=0;
    }
    streams=(hipStream_t *) malloc(n_streams*sizeof(hipStream_t)); //Allocates streams.
    for(unsigned int i=0; i< n_streams;i++)
        hipStreamCreate(&streams[i]);
    // Batch processing with streams
    unsigned int curr_str=0; //Stream to send each batch
    for (unsigned int i=0; i< num_particles/batch_size ;i++)
    {
        unsigned long long offset= i*streamSize;
        //Some division might give problems where num particles%nstreams != 0 . 
        hipMemcpyAsync(&d_all_particles_ptr[offset],&all_particles_ptr[offset], streamBytes, hipMemcpyHostToDevice, streams[curr_str]);
        GPU_update<<<streamSize/BLOCK_SIZE+1,BLOCK_SIZE,0,streams[curr_str]>>>(d_all_particles_ptr,v,offset,streamSize,num_iterations);
        hipMemcpyAsync(&all_particles_ptr[offset],&d_all_particles_ptr[offset], streamBytes, hipMemcpyDeviceToHost, streams[curr_str]);
        curr_str=(curr_str+1)%n_streams; //After sending one batch, uses next stream for the next one. 
    }
    hipDeviceSynchronize();
    //Checks result 
    for (long i = 0; i < num_particles; i++) 
    {
		if (fabs(all_particles_ptr[i].x - 1) > 0.01 ||fabs(all_particles_ptr[i].y - 1) > 0.01 || fabs(all_particles_ptr[i].z - 1) > 0.01  ) {
			fprintf(stderr, "Computation failed at index %ld", i);
			fprintf(stderr, "Values here: %f %f %f %f ", all_particles_ptr[i].x,all_particles_ptr[i].y,all_particles_ptr[i].z,all_particles_ptr[i+1].x);
			exit(1);
		}
    }
    printf("Output without errors!");
    //Destroys
    hipFree(d_all_particles_ptr);
    for(unsigned int i=0; i< n_streams;i++)
        hipStreamDestroy(streams[i]);
}


__global__ void GPU_update(Particle *d_all_particles_ptr, float v, unsigned long long offset, unsigned int streamSize,unsigned int num_iterations)
{
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < streamSize) //Stream size may not be a multiple of the block size used.
    {
        for(unsigned int j=0; j<num_iterations;j++)
	{
        unsigned long long curr_particle=i+offset; 
        float update_value=v*FIXED_DT;
        float3 update = make_float3(update_value, update_value, update_value);
        d_all_particles_ptr[curr_particle]+=update;
	}
    }

}
__device__ inline float3& operator +=(float3 &a, const float3 &b) { //https://forums.developer.nvidia.com/t/operator-overloading-for-float4/27228

    a.x += b.x;
    a.y += b.y;
    a.z += b.z;
    return a;
}
