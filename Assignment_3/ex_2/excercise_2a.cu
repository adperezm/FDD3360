#include "hip/hip_runtime.h"


// Question 1 defines:
#define FIXED_DT 1
#define BLOCK_SIZE 64
#define VEL 1
typedef float3 Particle;


//function declarations
__global__ void GPU_update(Particle *all_particles_ptr, float v,unsigned int num_particles); //Computes the update of only one update for all particles.
void particle_computation_launcher(bool use_cuda_malloc, unsigned int num_particles);
//function main
int main(int argc, char *argv[])
{
    printf("TBD");   
    unsigned int mode = atoi(argv[1]); //1 indicates cuda Malloc host, the other the simpler malloc.
    unsigned int num_particles = atoi(argv[2]);
    return 0;
}

//function definitions
void particle_computation_launcher(bool use_cuda_malloc, unsigned int num_particles)
{
    float v=VEL; //Could be a random value.
    Particle *all_particles_ptr = (Particle *) calloc(num_particles, sizeof(Particle)); 
    Particle *d_all_particles_ptr;
    if (use_cuda_malloc) //Allocation method
        hipMalloc(&d_all_particles_ptr, num_particles*sizeof(Particle));
    else
        hipHostMalloc(&d_all_particles_ptr, num_particles*sizeof(Particle));
    GPU_update<<<num_particles/block_size+1,block_size>>>(d_all_particles_ptr,v,num_iterations,num_particles);
    hipDeviceSynchronize();
    hipMemcpy(all_particles_ptr,d_all_particles_ptr,num_particles*sizeof(Particle),hipMemcpyDeviceToHost);
    hipFree(d_all_particles_ptr);

}


__global__ void GPU_update(Particle *all_particles_ptr, float v,unsigned int num_particles)
{
        const int i = blockIdx.x*blockDim.x + threadIdx.x;
        unsigned int curr_particle=i; //Check boundaries
        if(i<num_particles)
        {
            float update_value=v*FIXED_DT;
            float3 update = make_float3(update_value, update_value, update_value);
            all_particles_ptr[curr_particle]+=update;
        }
}