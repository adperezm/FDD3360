
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Question 1 defines:
#define FIXED_DT 1
#define BLOCK_SIZE 64
#define VEL 1
typedef float3 Particle;


//function declarations
__device__ inline float3& operator +=(float3 &a, const float3 &b);
__global__ void GPU_update(Particle *all_particles_ptr, float v,unsigned int num_particles); //Computes the update of only one update for all particles.
void particle_computation_launcher(bool use_cuda_malloc, unsigned int num_particles);
//function main
int main(int argc, char *argv[])
{
    printf("Running! \n");   
    unsigned int mode = atoi(argv[1]); //1 indicates cuda Malloc host, the other the simpler malloc.
    unsigned int num_particles = atoi(argv[2]);
    particle_computation_launcher(mode, num_particles); 

    return 0;
}

//function definitions
void particle_computation_launcher(bool use_cuda_malloc, unsigned int num_particles)
{
    float v=VEL; //Could be a random value.
    Particle *d_all_particles_ptr;
    Particle *all_particles_ptr;
    hipMalloc(&d_all_particles_ptr, num_particles*sizeof(Particle)); 
    if (use_cuda_malloc) //Allocation method
        all_particles_ptr = (Particle *) malloc(num_particles*sizeof(Particle));
    else
    {    
        //printf("Got here"); //cudamallochost is the same as cudahostalloc in this case https://stackoverflow.com/questions/35535831/is-there-any-difference-between-cudamallochost-and-cudahostalloc-without-spe#:~:text=cudaMallocHost()%20%3A%20%22Allocates%20page%2D,locked%20memory%20on%20the%20host.%22
        if (hipHostMalloc(&all_particles_ptr, num_particles*sizeof(Particle), hipHostMallocDefault) != hipSuccess)
	    {printf("Could not allocate pag memory!");}
    }
    //printf("What happened");
    for(unsigned int i=0; i< num_particles;i++)
    {
	all_particles_ptr[i].x=0;
	all_particles_ptr[i].y=0;
	all_particles_ptr[i].z=0;
    }
    hipMemcpy(d_all_particles_ptr,all_particles_ptr,num_particles*sizeof(Particle),hipMemcpyHostToDevice);
    GPU_update<<<num_particles/BLOCK_SIZE+1,BLOCK_SIZE>>>(d_all_particles_ptr,v,num_particles);
    hipDeviceSynchronize();
    hipMemcpy(all_particles_ptr,d_all_particles_ptr,num_particles*sizeof(Particle),hipMemcpyDeviceToHost);
    hipFree(d_all_particles_ptr);
    if (use_cuda_malloc)
        free(all_particles_ptr);
    else
        hipHostFree(all_particles_ptr);
}


__global__ void GPU_update(Particle *all_particles_ptr, float v,unsigned int num_particles)
{
        const int i = blockIdx.x*blockDim.x + threadIdx.x;
        unsigned int curr_particle=i; //Check boundaries
        if(i<num_particles)
        {
            float update_value=v*FIXED_DT;
            float3 update = make_float3(update_value, update_value, update_value);
            all_particles_ptr[curr_particle]+=update;
        }
}
__device__ inline float3& operator +=(float3 &a, const float3 &b) { //https://forums.developer.nvidia.com/t/operator-overloading-for-float4/27228

  a.x += b.x;
  a.y += b.y;
  a.z += b.z;
  return a;
}
