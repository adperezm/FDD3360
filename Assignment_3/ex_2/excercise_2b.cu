
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Question 1 defines:
#define FIXED_DT 1
#define BLOCK_SIZE 64
#define VEL 1
typedef float3 Particle;


//function declarations
__device__ inline float3& operator +=(float3 &a, const float3 &b);
__global__ void GPU_update(Particle *all_particles_ptr, float v,unsigned int num_particles); //Computes the update of only one update for all particles.
void particle_computation_launcher( unsigned int num_particles);
//function main
int main(int argc, char *argv[])
{
    printf("Running! \n");   
    unsigned int num_particles = atoi(argv[1]);
    particle_computation_launcher( num_particles); 

    return 0;
}

//function definitions
void particle_computation_launcher(unsigned int num_particles)
{
    float v=VEL; //Could be a random value.
    Particle *all_particles_ptr;
    if(hipMallocManaged(&all_particles_ptr, num_particles*sizeof(Particle),hipMemAttachGlobal)!= hipSuccess)
        printf("Could not allocate pag memory!");
    
    //printf("What happened");
    for(unsigned int i=0; i< num_particles;i++)
    {
	all_particles_ptr[i].x=0;
	all_particles_ptr[i].y=0;
	all_particles_ptr[i].z=0;
    }
    GPU_update<<<num_particles/BLOCK_SIZE+1,BLOCK_SIZE>>>(all_particles_ptr,v,num_particles);
    hipDeviceSynchronize();
    hipFree(all_particles_ptr);
}


__global__ void GPU_update(Particle *all_particles_ptr, float v,unsigned int num_particles)
{
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int curr_particle=i; //Check boundaries
    if(i<num_particles)
    {
        float update_value=v*FIXED_DT;
        float3 update = make_float3(update_value, update_value, update_value);
        all_particles_ptr[curr_particle]+=update;
    }
}
__device__ inline float3& operator +=(float3 &a, const float3 &b) { //https://forums.developer.nvidia.com/t/operator-overloading-for-float4/27228

  a.x += b.x;
  a.y += b.y;
  a.z += b.z;
  return a;
}
