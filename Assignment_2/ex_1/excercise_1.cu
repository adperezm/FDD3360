
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 1
#define TPB 256



__global__ void printHelloWorld()
{
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	__syncthreads();
	printf("Hello world! My threadId is  %2d \n",i);
}

int main()
{
  printHelloWorld<<<N, TPB>>>();
  hipDeviceSynchronize();
  return 0;
}
