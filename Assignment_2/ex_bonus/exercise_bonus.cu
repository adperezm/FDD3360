
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>
#include <sys/time.h>

#define NUM_ITER 1000000000
#define TPB 128
#define NB 1

double cpuSecond() {
  struct timeval tp;
  gettimeofday(&tp,NULL);
  return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

__global__ void gpu_random(float *res,hiprandState *states) {

  int id = blockIdx.x*blockDim.x+threadIdx.x;
  double x, y, z;
  float count=0;

  int seed=id; //different seed per thread
  hiprand_init(seed,id,0,&states[id]); //initialize curand

  for (int i=0;i < NUM_ITER/(NB*TPB); i++){
    //Generate a random point
    x=hiprand_uniform_double(&states[id]);
    y=hiprand_uniform_double(&states[id]);
    // Check if point is in unit circle
    z = sqrt((x*x) + (y*y));
    if (z <= 1.0){
      count=count+1;
    }
  }
  //Store the values
  res[id]=count;

}



int main(int argc, char* argv[])
{
    float count=0;
    double pi;

    //Define an array of size (num of threads) to hold the results in GPU
    float *d_x=0;
    hipMalloc(&d_x,NB*TPB*sizeof(float));

    // Hold the results in CPU
    float *cpu_x= (float *)malloc(NB*TPB*sizeof(float));

    //Initialize randon numbers in GPU
    hiprandState *dev_random;
    hipMalloc((void**)&dev_random,NB*TPB*sizeof(hiprandState));

    double iStart=cpuSecond();
    //run the kernel
    gpu_random <<<NB,TPB>>> (d_x,dev_random);
    hipDeviceSynchronize();
 
    //copy back into CPU
    hipMemcpy(cpu_x,d_x,NB*TPB*sizeof(float),hipMemcpyDeviceToHost);
    hipFree(d_x);

    for (int i=0;i< NB*TPB; i++){
    count = count + cpu_x[i];
    }
    
    // Estimate Pi and display the result
    pi = ((double)count / (double)NUM_ITER) * 4.0;
    
    double iElaps=cpuSecond() - iStart;

    //for (int i=0;i< NB*TPB; i++){
    //  printf("x = %f\n", cpu_x[i]);
    //}


    printf("The result is %f\n", pi);
    printf("time elapsed =  %f seconds\n", iElaps);


    return 0;
}


