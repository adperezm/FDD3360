
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>
#include <sys/time.h>

#define SINGLE_PRECISION


double cpuSecond() {
  struct timeval tp;
  gettimeofday(&tp,NULL);
  return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

__global__ void gpu_random(float *res,hiprandState *states,unsigned int blockSize,unsigned long long numIter,unsigned int n_blocks) {

  int id = blockIdx.x*blockDim.x+threadIdx.x;
#ifdef SINGLE_PRECISION
  float x, y, z;
#else
  double x, y, z;
#endif
  float count=0;

  int seed=id; //different seed per thread
  hiprand_init(seed,id,0,&states[id]); //initialize curand

  for (int i=0;i < (numIter/(n_blocks*blockSize)+1); i++){ //+1 because might give 0 that division
    //Generate a random point
#ifdef SINGLE_PRECISION
    x=hiprand_uniform(&states[id]);
    y=hiprand_uniform(&states[id]);
#else
    x=curand_uniform_double(&states[id]);
    y=curand_uniform_double(&states[id]);
#endif
    // Check if point is in unit circle
    z = sqrt((x*x) + (y*y));
    if (z <= 1.0){
      count=count+1;
    }
  }
  //Store the values
  res[id]=count;

}



int main(int argc, char* argv[])
{
    float count=0;
    double pi;
    unsigned long long numIter;
    unsigned int blockSize= atoi(argv[1]);
    unsigned int n_blocks= 1000000/blockSize; // So that there are always a million threads
    //Define an array of size (num of threads) to hold the results in GPU
    float *d_x=0;
    scanf("%llu", &numIter);
    numIter=(((unsigned long long)1000000)*numIter);
    hipMalloc(&d_x,n_blocks*blockSize*sizeof(float));
    printf("Num iter %llu, Block size: %d, N blocks: %d\n", numIter, blockSize, n_blocks);
    // Hold the results in CPU
    float *cpu_x= (float *)malloc(n_blocks*blockSize*sizeof(float));



    //Initialize randon numbers in GPU
    hiprandState *dev_random;
    hipMalloc((void**)&dev_random,n_blocks*blockSize*sizeof(hiprandState));

    double iStart=cpuSecond();
    //run the kernel
    gpu_random <<<n_blocks,blockSize>>> (d_x,dev_random,blockSize,numIter,n_blocks);
    hipDeviceSynchronize();
 
    //copy back into CPU
    hipMemcpy(cpu_x,d_x,n_blocks*blockSize*sizeof(float),hipMemcpyDeviceToHost);
    hipFree(d_x);

    for (int i=0;i< n_blocks*blockSize; i++){
    count = count + cpu_x[i];
    }
    
    // Estimate Pi and display the result
    pi = ((double)count / (double)numIter) * 4.0;
    
    double iElaps=cpuSecond() - iStart;

    //for (int i=0;i< NB*TPB; i++){
    //  printf("x = %f\n", cpu_x[i]);
    //}


    printf("The result is %f\n", pi);
    printf("time elapsed =  %f seconds\n", iElaps);


    return 0;
}


