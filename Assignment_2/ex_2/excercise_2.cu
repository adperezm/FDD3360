
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>


#define TPB 256


double cpuSecond() {
   struct timeval tp;
   gettimeofday(&tp,NULL);
   return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

__global__ void SAXPY(float *out, float a, float * x, float * y,unsigned int arraySize)
{
        const int i = blockIdx.x*blockDim.x + threadIdx.x;
        if(i<arraySize)
        {
                out[i]=x[i]*a+y[i];
        }
}
void SAXPYLauncher(float *out, float a, float * x, float * y,unsigned int arraySize) {
  float *d_out;
  float *d_x;
  float *d_y;

  hipMalloc(&d_out, arraySize*sizeof(float));
  hipMalloc(&d_x, arraySize*sizeof(float));
  hipMalloc(&d_y, arraySize*sizeof(float));

  hipMemcpy(d_x, x, arraySize*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, arraySize*sizeof(float), hipMemcpyHostToDevice);

  double iStart = cpuSecond();
  SAXPY<<<arraySize/TPB + 1, TPB>>>(d_out, a, d_x, d_y,arraySize);
  hipDeviceSynchronize();
  double calc_time = cpuSecond() - iStart;
  hipMemcpy(out, d_out, arraySize*sizeof(float), hipMemcpyDeviceToHost);

  printf("GPU calculation time: %f  ", calc_time);
  hipFree(d_out);
  hipFree(d_x);
  hipFree(d_y);
}

void CPU_saxpy(float *out, float a, float * x, float * y,unsigned int arraySize) {
        for(int j = 0; j < arraySize; j++)
        {
                out[j]=x[j]*a+y[j];
		}
}

int main(int argc, char *argv[])
{
        unsigned int arraySize= atoi(argv[1]);
        float *x = (float *)malloc(arraySize*sizeof(float));
        float *y = (float *)malloc(arraySize*sizeof(float));
        float *out_gpu = (float *)malloc(arraySize*sizeof(float));
        float *out_cpu = (float *)malloc(arraySize*sizeof(float));
        float a=3;
        float den=0;
        for(int i=0; i< arraySize;i++)
		{
                x[i]=1;
                y[i]=3;
        }
        printf("Array size: %d\n", arraySize);
        printf("Computing SAXPY on the CPU…");
        double iStart = cpuSecond();
        CPU_saxpy(out_cpu, a, x, y,arraySize);
        double iElaps_CPU = cpuSecond() - iStart;
        printf("Done! \n");
        printf("Computing SAXPY on the GPU…");
        iStart = cpuSecond();
        SAXPYLauncher(out_gpu, a, x, y,arraySize);
        double iElaps_GPU = cpuSecond() - iStart;
        printf("Done! \n");
        printf("Time elapsed CPU: %f \n Time elapsed GPU: %f \n",iElaps_CPU,iElaps_GPU);
        printf("First elements out array: %f, %f, %f",out_cpu[0],out_cpu[1],out_cpu[2]);
        for(int i = 0; i < arraySize; i++)
        {
                if (abs(out_cpu[i])< 1e-6)
                {       den=1e-6; }
                else
                {       den = out_cpu[i]; }

                if(abs(out_gpu[i]-out_cpu[i])/den> 1e-3) {
                        printf("Difference encountered! \n");
                        break;
                }
        }

        return 0;
}

		